#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>
#include <torch/types.h>
#include <vector>
#include <hipcub/hipcub.hpp>

#define WARP_SIZE 32
#define BLOCK_DIM 256
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])

// -------------------------------------- FP32
// -------------------------------------- Warp Reduce Sum
template <const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f32(float val) {
#pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return val;
}

// Block reduce sum device helper for RMS Norm
template <const int NUM_THREADS = 256>
__device__ __forceinline__ float block_reduce_sum_f32(float val) {
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;
  static __shared__ float shared[NUM_WARPS];

  val = warp_reduce_sum_f32<WARP_SIZE>(val);
  if (lane == 0)
    shared[warp] = val;
  __syncthreads();
  val = (lane < NUM_WARPS) ? shared[lane] : 0.0f;
  val = warp_reduce_sum_f32<NUM_WARPS>(val);
  return val;
}

// RMS Norm: x: NxK, y': NxK, y'=x/rms(x) each row
template <const int NUM_THREADS = 256>
__global__ void rms_norm_f32_kernel(float *x, float *y, float g, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x;  // 0..N-1
  int idx = bid * blockDim.x + threadIdx.x;
  const float epsilon = 1e-5f;

  __shared__ float s_variance;                 // shared within block
  float value = (idx < N * K) ? x[idx] : 0.0f; // load once only
  float variance = value * value;
  variance = block_reduce_sum_f32<NUM_THREADS>(variance);
  if (tid == 0)
    s_variance = rsqrtf(variance / (float)K + epsilon);
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads();
  if (idx < N * K)
    y[idx] = (value * s_variance) * g;
}

// RMS Norm Vec4: x: NxK, y': NxK, y'=x/rms(x) each row
template <const int NUM_THREADS = 256 / 4>
__global__ void rms_norm_f32x4_kernel(float *x, float *y, float g, int N,
                                      int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x;  // 0..N-1
  int idx = (bid * blockDim.x + threadIdx.x) * 4;
  const float epsilon = 1e-5f;

  __shared__ float s_variance; // shared within block
  float4 reg_x = FLOAT4(x[idx]);
  float variance = (idx < N * K) ? (reg_x.x * reg_x.x + reg_x.y * reg_x.y +
                                    reg_x.z * reg_x.z + reg_x.w * reg_x.w)
                                 : 0.0f;
  variance = block_reduce_sum_f32<NUM_THREADS>(variance);
  if (tid == 0)
    s_variance = rsqrtf(variance / (float)K + epsilon);
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads();
  float4 reg_y;
  reg_y.x = reg_x.x * s_variance * g;
  reg_y.y = reg_x.y * s_variance * g;
  reg_y.z = reg_x.z * s_variance * g;
  reg_y.w = reg_x.w * s_variance * g;
  if (idx < N * K)
    FLOAT4(y[idx]) = reg_y;
}

// 新增的row RMS norm核函数
__global__ void row_rmsnorm_f32(float *x, float g, float *y, int K) {
  const float epsilon = 1e-5f;
  const int tid = threadIdx.x;

  constexpr int pack_size = 4;
  const int pack_num = K / pack_size;
  const int pack_off = pack_size * pack_num;

  float variance = 0.0f;
  float4 *x_pack = reinterpret_cast<float4 *>(x);
  for (int i = tid; i < pack_num; i += blockDim.x) {
    float4 x_float4 = *(x_pack + i);
    variance += x_float4.x * x_float4.x;
    variance += x_float4.y * x_float4.y;
    variance += x_float4.z * x_float4.z;
    variance += x_float4.w * x_float4.w;
  }

  for (int i = pack_off + tid; i < K; i += blockDim.x) {
    variance += x[i] * x[i];
  }

  using BlockReduce = hipcub::BlockReduce<float, BLOCK_DIM>;
  __shared__ typename BlockReduce::TempStorage temp;
  __shared__ float s_variance;
  variance = BlockReduce(temp).Sum(variance);
  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / static_cast<float>(K) + epsilon);
  }
  __syncthreads();

  float4 *y_pack = reinterpret_cast<float4 *>(y);
  for (int i = tid; i < pack_num; i += blockDim.x) {
    float4 x_float4 = *(x_pack + i);
    *(y_pack + i) = make_float4(s_variance * x_float4.x * g,
                                s_variance * x_float4.y * g,
                                s_variance * x_float4.z * g,
                                s_variance * x_float4.w * g);
  }

  for (int i = pack_off + tid; i < K; i += blockDim.x) {
    y[i] = g * x[i] * s_variance;
  }
}

// --------------------- PyTorch bindings for custom kernel -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)                                   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                                   \
  if (((T).options().dtype() != (th_type))) {                                  \
    std::cout << "Tensor Info:" << (T).options() << std::endl;                 \
    throw std::runtime_error("values must be " #th_type);                      \
  }

#define CHECK_TORCH_TENSOR_SHAPE(T1, T2)                                       \
  assert((T1).dim() == (T2).dim());                                            \
  for (int i = 0; i < (T1).dim(); ++i) {                                       \
    if ((T2).size(i) != (T1).size(i)) {                                        \
      throw std::runtime_error("Tensor size mismatch!");                       \
    }                                                                          \
  }

#define LANUCH_RMS_NORM_F32_KERNEL(K)                                          \
  rms_norm_f32_kernel<(K)>                                                     \
      <<<grid, block>>>(reinterpret_cast<float *>(x.data_ptr()),               \
                        reinterpret_cast<float *>(y.data_ptr()), g, N, (K));

#define DISPATCH_RMS_NORM_F32_KERNEL(N, K)                                     \
  dim3 block((K));                                                             \
  dim3 grid((N));                                                              \
                                                                               \
  switch ((K)) {                                                               \
  case 64:                                                                     \
    LANUCH_RMS_NORM_F32_KERNEL(64)                                             \
    break;                                                                     \
  case 128:                                                                    \
    LANUCH_RMS_NORM_F32_KERNEL(128)                                            \
    break;                                                                     \
  case 256:                                                                    \
    LANUCH_RMS_NORM_F32_KERNEL(256)                                            \
    break;                                                                     \
  case 512:                                                                    \
    LANUCH_RMS_NORM_F32_KERNEL(512)                                            \
    break;                                                                     \
  case 768:                                                                    \
    LANUCH_RMS_NORM_F32_KERNEL(768)                                            \
    break;                                                                     \
  case 1024:                                                                   \
    LANUCH_RMS_NORM_F32_KERNEL(1024)                                           \
    break;                                                                     \
  default:                                                                     \
    if ((K) <= 1024) {                                                         \
      rms_norm_f32_kernel<1024>                                                \
          <<<grid, block>>>(reinterpret_cast<float *>(x.data_ptr()),           \
                            reinterpret_cast<float *>(y.data_ptr()), g, N, (K)); \
    } else {                                                                    \
      throw std::runtime_error("K too large for F32 kernel, use F32x4 instead"); \
    }                                                                          \
    break;                                                                     \
  }

#define LANUCH_RMS_NORM_F32x4_KERNEL(K)                                        \
  rms_norm_f32x4_kernel<(K) / 4>                                               \
      <<<grid, block>>>(reinterpret_cast<float *>(x.data_ptr()),               \
                        reinterpret_cast<float *>(y.data_ptr()), g, N, (K));

#define DISPATCH_RMS_NORM_F32x4_KERNEL(N, K)                                   \
  dim3 block((K) / 4);                                                         \
  dim3 grid((N));                                                              \
                                                                               \
  switch ((K)) {                                                               \
  case 64:                                                                     \
    LANUCH_RMS_NORM_F32x4_KERNEL(64) break;                                    \
  case 128:                                                                    \
    LANUCH_RMS_NORM_F32x4_KERNEL(128) break;                                   \
  case 256:                                                                    \
    LANUCH_RMS_NORM_F32x4_KERNEL(256) break;                                   \
  case 512:                                                                    \
    LANUCH_RMS_NORM_F32x4_KERNEL(512) break;                                   \
  case 768:                                                                    \
    LANUCH_RMS_NORM_F32x4_KERNEL(768) break;                                   \
  case 1024:                                                                   \
    LANUCH_RMS_NORM_F32x4_KERNEL(1024) break;                                  \
  case 1280:                                                                   \
    LANUCH_RMS_NORM_F32x4_KERNEL(1280) break;                                  \
  case 1536:                                                                   \
    LANUCH_RMS_NORM_F32x4_KERNEL(1536) break;                                  \
  case 1792:                                                                   \
    LANUCH_RMS_NORM_F32x4_KERNEL(1792) break;                                  \
  case 2048:                                                                   \
    LANUCH_RMS_NORM_F32x4_KERNEL(2048) break;                                  \
  case 2304:                                                                   \
    LANUCH_RMS_NORM_F32x4_KERNEL(2304) break;                                  \
  case 2560:                                                                   \
    LANUCH_RMS_NORM_F32x4_KERNEL(2560) break;                                  \
  case 2816:                                                                   \
    LANUCH_RMS_NORM_F32x4_KERNEL(2816) break;                                  \
  case 3072:                                                                   \
    LANUCH_RMS_NORM_F32x4_KERNEL(3072) break;                                  \
  case 3328:                                                                   \
    LANUCH_RMS_NORM_F32x4_KERNEL(3328) break;                                  \
  case 3584:                                                                   \
    LANUCH_RMS_NORM_F32x4_KERNEL(3584) break;                                  \
  case 3840:                                                                   \
    LANUCH_RMS_NORM_F32x4_KERNEL(3840) break;                                  \
  case 4096:                                                                   \
    LANUCH_RMS_NORM_F32x4_KERNEL(4096) break;                                  \
  case 4352:                                                                   \
    LANUCH_RMS_NORM_F32x4_KERNEL(4352) break;                                  \
  case 4608:                                                                   \
    LANUCH_RMS_NORM_F32x4_KERNEL(4608) break;                                  \
  case 4864:                                                                   \
    LANUCH_RMS_NORM_F32x4_KERNEL(4864) break;                                  \
  case 5120:                                                                   \
    LANUCH_RMS_NORM_F32x4_KERNEL(5120) break;                                  \
  default:                                                                     \
    if ((K) % 4 == 0 && (K) / 4 <= 1024) {                                     \
      rms_norm_f32x4_kernel<1024>                                              \
          <<<grid, block>>>(reinterpret_cast<float *>(x.data_ptr()),           \
                            reinterpret_cast<float *>(y.data_ptr()), g, N, (K)); \
    } else {                                                                    \
      throw std::runtime_error("Unsupported K size for F32x4 kernel");         \
    }                                                                          \
    break;                                                                     \
  }

void rms_norm_f32(torch::Tensor x, torch::Tensor y, float g) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kFloat32)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_RMS_NORM_F32_KERNEL(N, K)
}

void rms_norm_f32x4(torch::Tensor x, torch::Tensor y, float g) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kFloat32)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_RMS_NORM_F32x4_KERNEL(N, K)
}

// 新增PyTorch绑定函数
void row_rms_norm_f32(torch::Tensor x, torch::Tensor y, float g) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kFloat32)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  
  dim3 block(BLOCK_DIM);
  dim3 grid(N);
  
  for (int i = 0; i < N; i++) {
    row_rmsnorm_f32<<<1, block>>>(
      reinterpret_cast<float *>(x.data_ptr()) + i * K,
      g,
      reinterpret_cast<float *>(y.data_ptr()) + i * K,
      K
    );
  }
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  TORCH_BINDING_COMMON_EXTENSION(rms_norm_f32)
  TORCH_BINDING_COMMON_EXTENSION(rms_norm_f32x4)
  TORCH_BINDING_COMMON_EXTENSION(row_rms_norm_f32)
}